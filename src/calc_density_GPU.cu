#include "hip/hip_runtime.h"
#include "calc_density_GPU.cuh"

#define MIN(X, Y) (((X) < (Y)) ? (X) : (Y))

// compute clifford attractor points via GPU
__global__
void calcDensity_GPU(double x_min, double x_max, double y_min, double y_max, 
                    double* x, double* y, int N, int M, int* A) {
    int point_id = blockDim.x*blockIdx.x + threadIdx.x;
    int tmp_col = -1;
    int tmp_row = -1;
    double cell_width, cell_height;
    if (point_id < N) {
        // save dimensions of cell
        cell_width = (x_max - x_min)/(double)M;
        cell_height = (y_max - y_min)/(double)M;
        // find the cell in which the point falls
        tmp_col = MIN((x[point_id] - x_min)/cell_width, M-1);
        tmp_row = MIN((y[point_id] - y_min)/cell_height, M-1);
        // atomically increment cell count
        if (tmp_col >= 0 && tmp_row >= 0) { atomicAdd(&A[tmp_row*M + tmp_col], 1); }
    }
}

// namespace Wrapper {
    void device_wrapper(double x_min, double x_max, double y_min, double y_max, 
                        int NUM_POINTS, double* host_x, double* host_y, unsigned int bytes_points, 
                        int SIZE_MATRIX, int* host_mat, unsigned int bytes_mat) {
        //======================================================
        // ALOCATE & INITIALISE DEVICE DATA
        //======================================================
        // booleans to check if cuda played nicely
        hipError_t result_x, result_y, result_mat, sync_check;
        // allocate space on device for vectors
        double *device_x;
        double *device_y;
        result_x = hipMalloc((void**)&device_x, bytes_points);
        result_y = hipMalloc((void**)&device_y, bytes_points);
        if ((result_x != 0) && (result_y != 0)) {
            std::cout << "\t> " << "ERROR: Failed to allocate device memory.\n";
            throw std::runtime_error("ERROR: Failed to allocate device memory.\n");
        } else { std::cout << "\t> " "Successfully allocated: " << bytes_points << " bytes of data on the device.\n"; }
        // copy host density matrix to device
        result_x = hipMemcpy(device_x, host_x, bytes_points, hipMemcpyHostToDevice);
        result_y = hipMemcpy(device_y, host_y, bytes_points, hipMemcpyHostToDevice);
        if ((result_x != 0) && (result_y != 0)) {
            std::cout << "\t> " "ERROR: Failed to copy data to device memory.\n";
            throw std::runtime_error("ERROR: Failed to copy data to device memory.\n");
        } else { std::cout << "\t> " "Successfully coppied data to device memory.\n"; }
        // allocate space on device for density matrix
        int *device_mat;
        result_mat = hipMalloc((void**)&device_mat, bytes_mat);
        if (result_mat != 0) {
            std::cout << "\t> " "ERROR: Failed to allocate device memory.\n";
            throw std::runtime_error("ERROR: Failed to allocate device memory.\n");
        } else { std::cout << "\t> " "Successfully allocated: " << bytes_mat << " bytes of data on the device.\n"; }
        // copy host density matrix to device
        result_mat = hipMemcpy(device_mat, host_mat, bytes_mat, hipMemcpyHostToDevice);
        if (result_mat != 0) {
            std::cout << "\t> " "ERROR: Failed to copy data to device memory.\n";
            throw std::runtime_error("ERROR: Failed to copy data to device memory.\n");
        } else { std::cout << "\t> " "Successfully coppied data to device memory.\n"; }

        //======================================================
        // RUN KERNEL ON DEVICE
        //======================================================
        // Number of threads in each thread block
        int threadsPerBlock = 32 * 4 * 4;
        // Number of thread blocks in grid
        int numBlocks = (NUM_POINTS + threadsPerBlock - 1) / threadsPerBlock;
        std::cout << "\t> " "Number of blocks: " << numBlocks << "\n";
        std::cout << "\t> " "Number of treads/block: " << threadsPerBlock << "\n";
        // Execute the kernel
        std::cout << "\t> " "Calculating density of points...\n";
        calcDensity_GPU<<<numBlocks, threadsPerBlock>>>(x_min, x_max, y_min, y_max, device_x, device_y, NUM_POINTS, SIZE_MATRIX, device_mat);
        sync_check = hipDeviceSynchronize();
        if (sync_check != 0) {
            std::cout << "\t> " "ERROR: Kernel failed with code " << sync_check << ".\n";
            throw std::runtime_error("ERROR: Kernel failed.\n");
        }

        //======================================================
        // COPY DEVICE DATA BACK TO HOST
        //======================================================
        // Copy density matrix back to host
        result_mat = hipMemcpy(host_mat, device_mat, bytes_mat, hipMemcpyDeviceToHost);
        if (result_mat != 0) {
            std::cout << "\t> " "ERROR: Failed to copy data to host memory.\n";
            throw std::runtime_error("ERROR: Failed to copy data to host memory.\n");
        } else { std::cout << "\t> " "Successfully coppied data to host memory.\n"; }

        // Release device memory
        hipFree(device_x);
        hipFree(device_y);
        hipFree(device_mat);
    }
// }
